#include "cuda.cuh"
#include "functors.cuh"

namespace {

}

void CudaLPA::CreateGpuGraph()
{
}

thrust::host_vector<int> CudaLPA::Calculate()
{
	printf("Kernel execution, begin...\n");
	auto result = thrust::host_vector<int>(inputVertices.size());
	thrust::sequence(thrust::host, result.begin(), result.end(), 0);
	printf("Kernel execution, end...\n");
	return result;
}
