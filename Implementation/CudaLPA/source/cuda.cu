#include "cuda.cuh"
#include "functors.cuh"

namespace {

}

void CudaLPA::CreateGpuGraph()
{
	printf("Creating gpu graph (naive), begin...\n");
	const auto verticesCount = inputVertices.size();
	const auto edgesCount = inputEdges.size();

	// create cpu arrays
	thrust::host_vector<int> vertices(verticesCount);
	thrust::host_vector<int> edges(edgesCount * 2);

	int currentEdge = 0;
	for (int vertex = 0; vertex < verticesCount; vertex++)
	{
		for (const auto& edge : inputEdges)
		{
			if (edge.first == vertex)
			{
				edges[currentEdge] = edge.second;
				currentEdge++;
			}
			else if (edge.second == vertex)
			{
				edges[currentEdge] = edge.first;
				currentEdge++;
			}
		}

		vertices[vertex] = currentEdge;
	}

	if (vertices[verticesCount - 1] != edgesCount * 2)
		printf("Error, filled less data than expected\n");

	// copy arrays to gpu
	d_vertices = vertices;
	d_edges = edges;

	printf("Creating gpu graph (naive), end...\n");
}

thrust::host_vector<int> CudaLPA::Calculate()
{
	printf("Kernel execution, begin...\n");
	auto result = thrust::host_vector<int>(inputVertices.size());
	thrust::sequence(thrust::host, result.begin(), result.end(), 0);
	printf("Kernel execution, end...\n");
	return result;
}
