#include "hip/hip_runtime.h"
#include "cuda.cuh"

namespace {

	constexpr int countersSize = 32;

	__global__ void PerformLpaStep(int* communities, int* communitiesBuf, int* vertices, int* edges, int verticesCount)
	{
		int idx = blockDim.x * blockIdx.x + threadIdx.x;

		int countersIdx[countersSize];
		int counters[countersSize];
		int counterCount = 0;

		if (idx < verticesCount)
		{
			int edgesBegin = 0;
			if (idx > 0)
				edgesBegin = vertices[idx - 1];

			int edgesEnd = vertices[idx];

			for (int i = edgesBegin; i < edgesEnd; i++)
			{
				int neigh = edges[i];
				bool found = false;

				for (int y = 0; y < counterCount; y++)
				{
					if (countersIdx[y] == neigh)
					{
						found = true;
						counters[y] += 1;
						break;
					}
				}

				if (!found && counterCount < countersSize)
				{
					countersIdx[counterCount] = neigh;
					counters[counterCount] = 1;
					counterCount++;
				}
			}

			if (counterCount > 0)
			{
				int max = 0;
				for (int y = 1; y < counterCount; y++)
					if (counters[y] > counters[max])
						max = y;

				communitiesBuf[idx] = countersIdx[max];
			}
		}
	}
}

void CudaLPA::CreateGpuGraph()
{
	printf("Creating gpu graph (naive), begin...\n");
	const auto verticesCount = inputVertices.size();
	const auto edgesCount = inputEdges.size();

	// create cpu arrays
	thrust::host_vector<int> vertices(verticesCount);
	thrust::host_vector<int> edges(edgesCount * 2);

	int currentEdge = 0;
	for (int vertex = 0; vertex < verticesCount; vertex++)
	{
		for (const auto& edge : inputEdges)
		{
			if (edge.first == vertex)
			{
				edges[currentEdge] = edge.second;
				currentEdge++;
			}
			else if (edge.second == vertex)
			{
				edges[currentEdge] = edge.first;
				currentEdge++;
			}
		}

		vertices[vertex] = currentEdge;
	}

	if (vertices[verticesCount - 1] != edgesCount * 2)
		printf("Error, filled less data than expected\n");

	// copy arrays to gpu
	d_vertices = vertices;
	d_edges = edges;

	d_communities = thrust::device_vector<int>(verticesCount);
	d_communities_buf = thrust::device_vector<int>(verticesCount);

	thrust::sequence(thrust::device, d_communities.begin(), d_communities.end());

	printf("Creating gpu graph (naive), end...\n");
}

thrust::host_vector<int> CudaLPA::Calculate()
{
	printf("Kernel execution, begin...\n");

	const auto verticesCount = inputVertices.size();
	constexpr int iterations = 10;

	constexpr int threadsPerBlock = 256;
	const int blocksPerGrid = (verticesCount + threadsPerBlock - 1) / threadsPerBlock;

	auto cptr = thrust::raw_pointer_cast(d_communities.data());
	auto cptrBuf = thrust::raw_pointer_cast(d_communities_buf.data());
	auto vptr = thrust::raw_pointer_cast(d_vertices.data());
	auto eptr = thrust::raw_pointer_cast(d_edges.data());

	for (int i = 0; i < iterations; i++)
	{
		if (i % 2 == 0)
		{
			PerformLpaStep << <blocksPerGrid, threadsPerBlock >> > (cptr, cptrBuf, vptr, eptr, verticesCount);
		}
		else
		{
			PerformLpaStep << <blocksPerGrid, threadsPerBlock >> > (cptrBuf, cptr, vptr, eptr, verticesCount);
		}

		hipDeviceSynchronize();
	}

	auto result = thrust::host_vector<int>(inputVertices.size());
	if (iterations % 2 == 0)
		thrust::copy(d_communities.begin(), d_communities.end(), result.begin());
	else
		thrust::copy(d_communities_buf.begin(), d_communities_buf.end(), result.begin());

	printf("Kernel execution, end...\n");
	return result;
}
