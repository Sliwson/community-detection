#include "cuda.cuh"
#include "functors.cuh"

namespace {

}

thrust::host_vector<int> CalculateLPA(thrust::host_vector<int> vertices, thrust::host_vector<thrust::pair<int, int>> edges)
{
	printf("Kernel execution, begin...\n");
	auto result = thrust::host_vector<int>(vertices.size());
	thrust::sequence(thrust::host, result.begin(), result.end(), 0);
	printf("Kernel execution, end...\n");
	return result;
}
